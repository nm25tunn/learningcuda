#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__
void resolveWeight(int8_t *A,int8_t *B,int8_t *C, int n, int wx){

	int k = blockIdx.x*blockDim.x+threadIdx.x;

	int off = k * n;

	int8_t temp = 0;
	int b = 0;

	if(k<wx){
		for(int i=0;i<n;i++){
			b = off+i;
			temp+=A[i]*B[b];
		}
		C[k]=temp;
	}


}

int main(){

	FILE *fout = fopen("out.txt","w");
	int T = 512;
	int nodes,weightsX,weightsY;

	for(int o = 1; o < 10000;o++){
		if(o%1000 == 0){
			printf("Starting run %d\n" ,o);
		}
		fprintf(fout,"%d",o);
		nodes = o;
		weightsX = o;
		weightsY = o;
		int8_t *node, *d_node, *weight, *d_weight,*out,*gout,*d_out;

		node = (int8_t *)malloc(nodes*sizeof(int8_t));
		weight = (int8_t *)malloc(weightsX*weightsY*sizeof(int8_t));
		out = (int8_t *)malloc(weightsX*sizeof(int8_t));
		gout = (int8_t *)malloc(weightsX*sizeof(int8_t));

		srand(time(0));

		//Set up matrix 1
		for(int i = 0;i<nodes;i++){
			node[i] = rand() % 3 - 1;
		}

		//Set up matrix 2
		for(int i = 0;i<(weightsX*weightsY);i++){
			weight[i] = rand() % 3 - 1;
		}

		//Set up matrices for results
		for(int i = 0;i<weightsX;i++){
			out[i] = 0;
			gout[i] = 0;
		}

		unsigned int sstart = clock();
		for(int i = 0; i<weightsX;i++){
			for(int j = 0; j<weightsY;j++){
				out[i]+=(node[j]*weight[(nodes*i)+j]);
			}
		}
		//printf("Sequential time taken in ms %li\n" ,(clock() - sstart));
		int seqtime = clock()-sstart;
		fprintf(fout,",%d",seqtime);

		//CUDA parallel code
		hipMalloc(&d_node,nodes*sizeof(int8_t));
		hipMalloc(&d_weight,weightsX*weightsY*sizeof(int8_t));
		hipMalloc(&d_out,weightsX*sizeof(int8_t));

		hipMemcpy(d_node,node,nodes*sizeof(int8_t),hipMemcpyHostToDevice);
		hipMemcpy(d_weight,weight,weightsX*weightsY*sizeof(int8_t),hipMemcpyHostToDevice);
		hipMemcpy(d_out,gout,weightsX*sizeof(int8_t),hipMemcpyHostToDevice);

		unsigned int pstart = clock();
		resolveWeight<<<weightsX+(T-1)/T,T>>>(d_node,d_weight,d_out,nodes,weightsX);
		hipDeviceSynchronize();
		//printf("Parallel time taken in ms %li\n" ,(clock() - pstart));
		int partime = clock()-pstart;
		fprintf(fout,",%d",partime);

		hipMemcpy(gout,d_out,weightsX*sizeof(int8_t),hipMemcpyDeviceToHost);


//		printf("Value at 3: %i\n",out[2]);
//		printf("Value on gpu at 3: %i\n",gout[2]);
//		printf("Value at 5120: %i\n",out[5119]);
//		printf("Value on gpu at 5120: %i\n",gout[5119]);
//		printf("Value at 10000: %i\n",out[9999]);
//		printf("Value on gpu at 10000: %i\n",gout[9999]);

		int err = 0;
		for(int i = 0; i < weightsX; i++){
			err += out[i]-gout[i];
		}
//		printf("Error with CUDA: %i\n",err);
		fprintf(fout,",%d\n",err);

		free(node);
		free(weight);
		free(out);
		free(gout);
		hipDeviceReset();
	}
	fclose(fout);
}
